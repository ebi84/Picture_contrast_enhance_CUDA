#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <opencv2/opencv.hpp>

#define MIN(x,y)  y ^ ((x ^ y) & -(x < y))  // calculating minimum of two unsigned integers without branching
#define MAX(x,y)  x ^ ((x ^ y) & -(x < y))  // calculating maximum of two unsigned integers without branching

#define LOG false  // for debugging

using namespace std;
using namespace cv;

__global__ void tmp(unsigned int* dev_min, unsigned int* dev_max, unsigned int num_channel);

__global__ void initialize(unsigned int* dev_min, unsigned int* dev_max, unsigned int num_channel);

template <unsigned int block_size>
__global__ void reduction(unsigned char* dev_vec, unsigned int* dev_min,
	unsigned int* dev_max, unsigned int size, unsigned int channel);

__global__ void enhance(unsigned char* dev_vec, unsigned int* dev_min, unsigned int* dev_max,
	unsigned int size, unsigned int num_channel);



int main()
{
	hipError_t cudaStatus;

	Mat img = imread("original.jpg");  // Mat object for loading the input image
	unsigned int size = img.total();  // size of each channel (row*col)
	unsigned int num_channel = img.channels();  // number of channels
	unsigned int tot_size = size * num_channel;
	Mat* BGR = new Mat[num_channel];  // Mat object for splitting channels
	split(img, BGR);
	
	unsigned char *dev_vec;  // device array containing pixel values
	unsigned int *dev_min, * dev_max; // device arrays for containing minimum/maximum values of channels
	unsigned char* vec = new unsigned char[tot_size];  // host array containing pixel values

	// copying pixel values of channels to vec
	for (unsigned int i = 0; i < num_channel; i++)
	{
		for (unsigned int j = 0; j < size; j++)
		{
			vec[i * size + j] = (unsigned char)BGR[i].data[j];
		}
	}

	cudaStatus = hipMalloc((void**)&dev_vec, tot_size);
	if (cudaStatus != hipSuccess) { cerr << "memory allocation for dev_vec failed"; goto Error1; }
	cudaStatus = hipMalloc((void**)&dev_min, num_channel * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) { cerr << "memory allocation for dev_min failed"; goto Error2; }
	cudaStatus = hipMalloc((void**)&dev_max, num_channel * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) { cerr << "memory allocation for dev_max failed"; goto Error3; }

	cudaStatus = hipMemcpy(dev_vec, vec, tot_size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { cerr << "memory copying to device failed"; goto Error3; }
	initialize << <1, 32 >> > (dev_min, dev_max, num_channel);  // initalizing min and max arrays
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { cerr << "initializing min/max arrays failed"; goto Error3; }

	unsigned int size_grid_reduction(2), size_block_reduction(128);

	// each stream is responsible for finding min/max for one channel
	const unsigned int num_streams = 3;  
	if (num_streams < num_channel) {
		cerr << "num_stream must be equal or greater than num_channel" << endl; goto Error3;
	}

	hipStream_t streams[num_streams];
	for (unsigned int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);
	}

	// reduction kernel
	for (unsigned int i = 0; i < num_channel; i++)
	{
		switch (size_block_reduction)
		{
		case 1024:
			reduction<1024> << <size_grid_reduction, size_block_reduction, size_block_reduction, streams[i] >> > 
				(&dev_vec[i*size], dev_min, dev_max, size, i); break;
		case 512:
			reduction<512> << <size_grid_reduction, size_block_reduction, size_block_reduction, streams[i] >> >
				(&dev_vec[i * size], dev_min, dev_max, size, i); break;
		case 256:
			reduction<256> << <size_grid_reduction, size_block_reduction, size_block_reduction, streams[i] >> >
				(&dev_vec[i * size], dev_min, dev_max, size, i); break;
		case 128:
			reduction<128> << <size_grid_reduction, size_block_reduction, size_block_reduction, streams[i] >> >
				(&dev_vec[i * size], dev_min, dev_max, size, i); break;
		case 64:
			reduction<64> << <size_grid_reduction, size_block_reduction, size_block_reduction, streams[i] >> >
				(&dev_vec[i * size], dev_min, dev_max, size, i); break;
		case 32:
			reduction<32> << <size_grid_reduction, size_block_reduction, size_block_reduction, streams[i] >> >
				(&dev_vec[i * size], dev_min, dev_max, size, i); break;
		}
	}
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { cerr << "reduction failed"; goto Error3; }

	if (LOG) tmp<<<1,32>>>(dev_min,  dev_max, num_channel);

	unsigned int size_grid_enhance(2), size_block_enhance(128);

	// enhancing pixel values
	enhance << <size_grid_enhance, size_block_enhance >> >
		(dev_vec, dev_min, dev_max, size, num_channel);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { cerr << "enhance"; goto Error3; }

	cudaStatus = hipMemcpy(vec, dev_vec, tot_size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { cerr << "memory copying to host failed"; goto Error3; }

	for (unsigned int i = 0; i < num_channel; i++)
	{
		for (unsigned int j = 0; j < size; j++)
		{
			 BGR[i].data[j] = (uint8_t)vec[i * size + j];
		}
	}

	// merging all channels to a single Mat object
	merge(BGR, num_channel, img);
	imwrite("enhanced.jpg", img);

	for (unsigned int i = 0; i < num_streams; i++) {
		hipStreamDestroy(streams[i]);
	}

Error3:
	hipFree(dev_max);
Error2:
	hipFree(dev_min);
Error1:
	hipFree(dev_vec);
	delete[] BGR;
	delete[] vec;

	return 0;
}




__global__ void initialize(unsigned int* dev_min, unsigned int* dev_max, unsigned int num_channel)
{
	unsigned int gid = threadIdx.x + blockDim.x * gridDim.x;
	while (gid < num_channel)
	{
		dev_min[gid] = 255u;
		dev_max[gid] = 0;
		gid += blockDim.x * gridDim.x;
	}

}





template <unsigned int block_size>
__global__ void reduction(unsigned char* dev_vec, unsigned int* dev_min,
	unsigned int* dev_max, unsigned int size, unsigned int channel)
{
	extern __shared__ unsigned char SHARED_MIN[];
	unsigned char* SHARED_MAX = &SHARED_MIN[block_size];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * block_size + tid;
	unsigned int grid_size = block_size * gridDim.x;
	SHARED_MIN[tid] = 255u;
	SHARED_MAX[tid] = 0;

	while (i < size)
	{
		SHARED_MIN[tid] = MIN(SHARED_MIN[tid], dev_vec[i]);
		SHARED_MAX[tid] = MAX(SHARED_MAX[tid], dev_vec[i]);
		i += grid_size;
	}
	__syncthreads();

	if (block_size >= 512)
	{
		if (tid < 256)
		{
			SHARED_MIN[tid] = MIN(SHARED_MIN[tid], SHARED_MIN[tid + 256]);
			SHARED_MAX[tid] = MAX(SHARED_MAX[tid], SHARED_MAX[tid + 256]);
		}
		__syncthreads();
	}

	if (block_size >= 256)
	{
		if (tid < 128)
		{
			SHARED_MIN[tid] = MIN(SHARED_MIN[tid], SHARED_MIN[tid + 128]);
			SHARED_MAX[tid] = MAX(SHARED_MAX[tid], SHARED_MAX[tid + 128]);
		}
		__syncthreads();
	}

	if (block_size >= 128)
	{
		if (tid < 64)
		{
			SHARED_MIN[tid] = MIN(SHARED_MIN[tid], SHARED_MIN[tid + 64]);
			SHARED_MAX[tid] = MAX(SHARED_MAX[tid], SHARED_MAX[tid + 64]);
		}
		__syncthreads();
	}

	if (block_size >= 64)
	{
		if (tid < 32)
		{
			SHARED_MIN[tid] = MIN(SHARED_MIN[tid], SHARED_MIN[tid + 32]);
			SHARED_MAX[tid] = MAX(SHARED_MAX[tid], SHARED_MAX[tid + 32]);
		}
		__syncthreads();
	}

	if (tid < 32)
	{
		SHARED_MIN[tid] = MIN(SHARED_MIN[tid], SHARED_MIN[tid + 16]);
		SHARED_MIN[tid] = MIN(SHARED_MIN[tid], SHARED_MIN[tid + 8]);
		SHARED_MIN[tid] = MIN(SHARED_MIN[tid], SHARED_MIN[tid + 4]);
		SHARED_MIN[tid] = MIN(SHARED_MIN[tid], SHARED_MIN[tid + 2]);
		SHARED_MIN[tid] = MIN(SHARED_MIN[tid], SHARED_MIN[tid + 1]);

		SHARED_MAX[tid] = MAX(SHARED_MAX[tid], SHARED_MAX[tid + 16]);
		SHARED_MAX[tid] = MAX(SHARED_MAX[tid], SHARED_MAX[tid + 8]);
		SHARED_MAX[tid] = MAX(SHARED_MAX[tid], SHARED_MAX[tid + 4]);
		SHARED_MAX[tid] = MAX(SHARED_MAX[tid], SHARED_MAX[tid + 2]);
		SHARED_MAX[tid] = MAX(SHARED_MAX[tid], SHARED_MAX[tid + 1]);
	}
	if (tid == 0)
	{
		atomicMin(&dev_min[channel], (unsigned int) SHARED_MIN[0]);
		atomicMax(&dev_max[channel], (unsigned int) SHARED_MAX[0]);
	}
}





__global__ void enhance(unsigned char* dev_vec, unsigned int* dev_min, unsigned int* dev_max,
	unsigned int size, unsigned int num_channel)
{
	unsigned int grid_size = blockDim.x * gridDim.x;
	for (unsigned int i = 0; i < num_channel; i++)
	{
		unsigned int gid = threadIdx.x + blockIdx.x * blockDim.x;
		unsigned char MIN_PIXEL = dev_min[i];
		unsigned char RANGE_PIXEL = (dev_max[i] - dev_min[i]);
		while (gid < size)
		{
			dev_vec[i * size + gid] = (dev_vec[i * size + gid] - MIN_PIXEL) / (float)RANGE_PIXEL * 255;
			gid += grid_size;
		}
	}
}



__global__ void tmp(unsigned int* dev_min, unsigned int* dev_max, unsigned int num_channel)
{
	unsigned int i = threadIdx.x;
	if (threadIdx.x < num_channel)
	{
		printf("min[%u]= %u,    max[%u]= %u\n", i, dev_min[i], i, dev_max[i]);
	}
}